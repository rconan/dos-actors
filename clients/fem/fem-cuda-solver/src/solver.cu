#include "hip/hip_runtime.h"
#include "solver.hpp"

// state equation`
__global__ void step_kernel(mode_state_space *mss, double *v, double *y,
                            int n) {
  int i;
  double x0, x1;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x0 = mss[i].x0;
    x1 = mss[i].x1;
    // x0 <- a0 x0 + a2 x1 + b2 vi
    mss[i].x0 = mss[i].a0 * x0 + mss[i].a2 * x1 + mss[i].b2 * v[i];
    // x1 <- a1 x0 + a3 x1 + b3 vi
    mss[i].x1 = mss[i].a1 * x0 + mss[i].a3 * x1 + mss[i].b3 * v[i];
    y[i] = mss[i].x0;
  }
}

void state_space::build(int n_mode_, mode_state_space *mss, int n_input_,
                        double *i2m, int n_output_, double *m2o) {

  n_mode = n_mode_;
  n_input = n_input_;
  n_output = n_output_;
  d_dcg = NULL;
  hipblasCreate(&handle);
  hipMalloc(&d_mss, n_mode * sizeof(mode_state_space));
  hipMemcpy(d_mss, mss, n_mode * sizeof(mode_state_space),
             hipMemcpyHostToDevice);
  hipMalloc(&d_i2m, n_mode * n_input * sizeof(double));
  hipMemcpy(d_i2m, i2m, n_mode * n_input * sizeof(double),
             hipMemcpyHostToDevice);
  hipMalloc(&d_m2o, n_mode * n_output * sizeof(double));
  hipMemcpy(d_m2o, m2o, n_mode * n_output * sizeof(double),
             hipMemcpyHostToDevice);
  hipMalloc(&d_u, n_input * sizeof(double));
  hipMalloc(&d_v, n_mode * sizeof(double));
  hipMalloc(&d_x0, n_mode * sizeof(double));
  hipMalloc(&d_y, n_output * sizeof(double));
}
void state_space::dc_gain_compensator(double *dcg) {
  hipMalloc(&d_dcg, n_output * n_input * sizeof(double));
  hipMemcpy(d_dcg, dcg, n_output * n_input * sizeof(double),
             hipMemcpyHostToDevice);
}
void state_space::free() {
  hipblasDestroy(handle);
  hipFree(d_mss);
  hipFree(d_i2m);
  hipFree(d_m2o);
  hipFree(d_u);
  hipFree(d_v);
  hipFree(d_x0);
  hipFree(d_y);
  if (d_dcg != NULL)
    hipFree(d_dcg);
}
void state_space::step(double *u, double *y) {
  double alpha = 1.0;
  double beta = 0.0;
  dim3 block(256); // or whatever block size you want
  dim3 grid((n_mode + block.x - 1) / block.x); // ceil(n/block.x)

  hipMemcpy(d_u, u, n_input * sizeof(double), hipMemcpyHostToDevice);
  // v = Bu
  hipblasDgemv(handle, HIPBLAS_OP_T, n_input, n_mode, &alpha, d_i2m, n_input, d_u,
              1, &beta, d_v, 1);

  // update state equation
  step_kernel<<<grid, block>>>(d_mss, d_v, d_x0, n_mode);

  // y = Cx0
  hipblasDgemv(handle, HIPBLAS_OP_N, n_output, n_mode, &alpha, d_m2o, n_output,
              d_x0, 1, &beta, d_y, 1);
  hipMemcpy(y, d_y, n_output * sizeof(double), hipMemcpyDeviceToHost);

  if (d_dcg != NULL) {
    beta = 1.0;
    hipblasDgemv(handle, HIPBLAS_OP_N, n_output, n_input, &alpha, d_dcg, n_output,
                d_u, 1, &beta, d_y, 1);
  }
  hipMemcpy(y, d_y, n_output * sizeof(double), hipMemcpyDeviceToHost);
}
